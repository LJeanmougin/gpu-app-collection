
#include <hip/hip_runtime.h>
#include <stdio.h>
#define BSIZE 1024
#define NUMBLOCK 1

__global__ void vecaddKernel(int *v1_in, int *v2_in, int *v_out, int size)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    v_out[idx] = v1_in[idx] + v2_in[idx];
}

int main()
{
    int h_vec1[NUMBLOCK * BSIZE];
    int h_vec2[NUMBLOCK * BSIZE];
    int h_out[NUMBLOCK * BSIZE];
    int *d_vec1, *d_vec2, *d_out;
    for(int i = 0; i < NUMBLOCK * BSIZE; i++)
    {
        h_vec1[i] = i;
        h_vec2[i] = i;
    }
    hipMalloc((void **) &d_vec1, NUMBLOCK * BSIZE * sizeof(int));
    hipMalloc((void **) &d_vec2, NUMBLOCK * BSIZE * sizeof(int));
    hipMalloc((void **) &d_out, NUMBLOCK * BSIZE * sizeof(int));
    hipMemcpy(d_vec1, h_vec1, NUMBLOCK * BSIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_vec2, h_vec2, NUMBLOCK * BSIZE * sizeof(int), hipMemcpyHostToDevice);
    vecaddKernel<<<NUMBLOCK, BSIZE>>>(d_vec1, d_vec2, d_out, NUMBLOCK * BSIZE);
    hipMemcpy(h_out, d_out, NUMBLOCK * BSIZE * sizeof(int), hipMemcpyDeviceToHost);
    return 0;
}